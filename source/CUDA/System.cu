#include "hip/hip_runtime.h"
#include <cutil.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>


//#include "Kernel.cu"
#include "radixsort.cuh"

//START OF KERNEL.CU!!!!!!!!! (END AT LINE 467) 

#pragma once

#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "Params.cuh"


typedef unsigned int uint;

#if USE_TEX
texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldVelTex;

texture<uint2, 1, hipReadModeElementType> particleHashTex;
texture<uint,  1, hipReadModeElementType> cellStartTex;

texture<float, 1, hipReadModeElementType> pressureTex;
texture<float, 1, hipReadModeElementType> densityTex;

texture<float, 1, hipReadModeElementType> dyeColorTex;
#endif

__constant__ SimParams par;



//----------------------------------------------------------------------------------------------------------------------------
///  Boundary Conditions
//----------------------------------------------------------------------------------------------------------------------------

__device__ void boundary(float3& pos, float3& vel)
{
	//  world box
	float3 wmin = par.worldMin, wmax = par.worldMax;

	float b = par.distBndSoft, stiff = par.bndStiff, damp = par.bndDamp, damp2 = par.bndDampC;
	float accBnd, diff;  float3 norm;
	BndType t = par.bndType;	bool bCylY = t==BND_CYL_Y, bCylZ = t==BND_CYL_Z,
		/*Eff*/ bWave = par.bndEffZ == BND_EFF_WAVE, bNoEff = par.bndEffZ == BND_EFF_NONE, bCycle = par.bndEffZ == BND_EFF_CYCLE;

	#define  EPS	0.00001f	// epsilon for collision detection
	#define  addB()  accBnd = stiff * diff - damp * dot(norm, vel);  vel += accBnd * norm * par.timeStep;	// box,pump, soft
	#define  addC()  accBnd = stiff * diff - damp2* dot(norm, vel);  vel += accBnd * norm * par.timeStep;	// cyl,sphr

	if (bWave)
	{	float sl = -par.r2Angle;
		//  slope
		diff = b - (pos.y - wmin.y) - (pos.z - wmin.z)*sl;	if (diff > EPS)  {  norm = make_float3(0, 1-sl, sl);  addB();  }
		//  shore waves
		wmin.z += par.rTwist*(1.f + sinf(par.rAngle));
	}

	//----------------  Box
	if (t != BND_SPHERE)
	{
		if (!bCylY)  {
			if (bNoEff || bWave){	diff = b - pos.z + wmin.z;	if (diff > EPS)  {	norm = make_float3( 0, 0, 1);  addC();  }	}
			if (!bCycle)	{		diff = b + pos.z - wmax.z;	if (diff > EPS)  {	norm = make_float3( 0, 0,-1);  addC();  }	}
		}
		if (!bCylY && !bCylZ)  {
			diff = b - pos.x + wmin.x;	if (diff > EPS)  {	norm = make_float3( 1, 0, 0);  addB();  }
			diff = b + pos.x - wmax.x;	if (diff > EPS)  {	norm = make_float3(-1, 0, 0);  addB();  }	}

		if (!bCylZ)  {
			diff = b - pos.y + wmin.y;	if (diff > EPS)  {	norm = make_float3( 0, 1, 0);  addB();  }
			diff = b + pos.y - wmax.y;	if (diff > EPS)  {	norm = make_float3( 0,-1, 0);  addB();  }	}
	}
	else	//  Sphere
	/*if (t == BND_SPHERE)*/  {
		float len = length(pos);	diff = b + len + wmin.y;
		if (diff > EPS)  {	norm = make_float3( -pos.x/len, -pos.y/len, -pos.z/len);  addC();  }  }

	//  Cylinder Y|
	if (bCylY || t == BND_CYL_YZ)  {
		float len = length(make_float2(pos.x, pos.z));		diff = b + len - wmax.x;
		if (diff > EPS)  {	norm = make_float3( -pos.x/len, 0, -pos.z/len);  addC();  }	}

	//  Cylinder Z--
	if (bCylZ || t == BND_CYL_YZ)  {
		float len = length(make_float2(pos.x, pos.y));		diff = b + len + wmin.y;
		if (diff > EPS)  {	norm = make_float3( -pos.x/len, -pos.y/len, 0);  addC();  }  }
	
	//  Wrap, Cycle  Z--
	if (!bWave && !bNoEff)  {		float dr = 1.f*par.particleR;/*par.rDexit*/
		if (bCycle &&
			vel.z > par.rVexit && pos.z > wmax.z -b -dr) {  pos.z -= wmax.z - wmin.z -2*b -dr;  }  else
		if (vel.z <-par.rVexit && pos.z < wmin.z +b +dr) {  pos.z += wmax.z - wmin.z -2*b -dr;  }  }


	///  Pump  ~~~~~~~~~~~~~~~~~~~~~~~~~~
	if (t == BND_PUMP_Y)
	{
		float rad = wmax.x, ang = par.angOut, hc = par.hClose, rin = rad * par.radIn;
		
		//  cyl Y|  Frame
		float len = length(make_float2(pos.x, pos.y));		diff = b + len - rad;
		if (diff > EPS)
		{	float a = atan(pos.x / pos.y);
			if (ang < 0.5f)
			{	if (a < -ang || a > ang || pos.y < 0)			// ^
				{	norm = make_float3( -pos.x/len, -pos.y/len, 0);  addB();  }	}
			else
			{	if (pos.y < 0 || (len < rad*par.s5 && a < ang))	// S
				{	norm = make_float3( -pos.x/len, -pos.y/len, 0);  addB();  }	}
		}
		
		//  box Out
		float xs;
		if (ang < 0.5f)		// ^
		{	xs = sinf(ang * par.s3)*rad;
			float zs = cosf(ang * par.s3)*rad * par.s4;
			if (pos.y > zs)
			{	diff = b - pos.x - xs;	if (diff > EPS)  {	norm = make_float3( 1, 0, 0);  addB();  }
				diff = b + pos.x - xs;	if (diff > EPS)  {	norm = make_float3(-1, 0, 0);  addB();  }
			}
		}else{	// S
			xs = 0.09f*par.s4;
			if (len >= rad*par.s6)
			{	diff = b - pos.x + xs;	if (diff > EPS)  {	norm = make_float3( 1, 0, 0);  addB();  }	}
		}
		
		//  cyl hole In  (upper close)
		if (pos.z > hc - b*par.s1)
		{	diff = b + len - rin;	if (diff > EPS)  {	norm = make_float3( -pos.x/len, -pos.y/len, 0);  addB();  }	}
		if (pos.z < hc - b*par.s2)
		{	diff = b + pos.z - hc;	if (diff > EPS)  {	norm = make_float3( 0, 0,-1);  addB();  }	}
		
		//  wrap out-in
		diff = pos.y - wmax.y + par.rDexit;
		if (diff > EPS && vel.y > par.rVexit)
		{
			float aa, rr;	// mapping rect-circle
			if (ang < 0.5f)
			{	float xx = xs*2, zz = abs(hc - wmin.z);
				rr = (pos.x + xx/2)/xx * 0.7f;
				aa = (pos.z - zz/2)/zz * 1.6f;
			} else {
				float xx = xs, zz = abs(hc - wmin.z);
				rr = (wmax.x - pos.x)/xs * 0.45f;
				aa = (pos.z - zz/2)/zz * 1.8f;
			}
			rr *= rin;  aa *= PI2;
			float x = cosf(aa) * rr, y = sinf(aa) * rr,
				  z = wmax.z -b - /**/abs(vel.y - par.rVexit)*0.01;
			pos = make_float3(x, y, z);
			vel = make_float3(vel.x,vel.z,-vel.y);
		}
	}	//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
}

///----------------------------------------------------------------------------------------------------------------------------
///  Integrate
///----------------------------------------------------------------------------------------------------------------------------

__global__ void integrateD(float4* newPos, float4* newVel, float4* oldPos, float4* oldVel)
{
	int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	float4 pos4 = oldPos[index];	float4 vel4 = oldVel[index];
	float3 pos = make_float3(pos4);	float3 vel = make_float3(vel4);


	boundary(pos, vel);  // bnd
	
	
	///  Euler integration  -------------------------------
	vel += par.gravity * par.timeStep;	// v(t) = a(t) dt
	vel *= par.globalDamping;	// = 1
	pos += vel * par.timeStep;			// p(t+1) = p(t) + v(t) dt

	/*accel += m_Gravity;  accel *= m_DT;
	vel += accel;				// v(t+1) = v(t) + a(t) dt
	vel_eval += accel;	vel_eval *= m_DT/d;
	pos += vel_eval;	vel_eval = vel;
	///  Leapfrog Integration  ----------------------------
	/*vnext = (accel + m_Gravity) * m_DT + vel;		// v(t+1/2) = v(t-1/2) + a(t) dt
	vel_eval = (vel + vnext) * 0.5f;	// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5		used to compute forces later
	vel = vnext;	vnext *= m_DT/d;
	pos += vnext;			// p(t+1) = p(t) + v(t+1/2) dt
	/**/
	
	//----------------  Hard boundary
	float b = par.distBndHard;
	float3 wmin = par.worldMin, wmax = par.worldMax;
	if (pos.x > wmax.x - b)   pos.x = wmax.x - b;
	if (pos.x < wmin.x + b)   pos.x = wmin.x + b;
	if (pos.y > wmax.y - b)   pos.y = wmax.y - b;
	if (pos.y < wmin.y + b)   pos.y = wmin.y + b;
	if (pos.z > wmax.z - b)   pos.z = wmax.z - b;
	if (pos.z < wmin.z + b)   pos.z = wmin.z + b;

	// store new position and velocity
	newPos[index] = make_float4(pos, pos4.w);
	newVel[index] = make_float4(vel, vel4.w);
}



#include "Kernel_Cell.cui"



//----------------------------------------------------------------------------------------------------------------------------
///  Compute SPH  Force
//----------------------------------------------------------------------------------------------------------------------------

__device__ float W(float x, float h){
	if (x<=h){
	return (1-x)/h;
	}
	else return 0.0;
}

__device__ float phiClamp(float I,float Tmin,float Tmax){
return (min(I,Tmax)-min(I,Tmin))/(Tmax-Tmin);
}

__global__ void computeForceD(float4* newPos, float4* newVel, float4* oldPos, float4* oldVel, 
		float4* clr, float* pressure, float* density, float* dyeColor/**/,	uint2* particleHash,  uint* cellStart)
{
	int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	float4 pos = FETCH(oldPos, index);		float4 vel = FETCH(oldVel, index);
	float pres = FETCH(pressure, index);	float dens = FETCH(density, index);
	int3 gridPos = calcGridPos(pos);

	float3 addVel = make_float3(0.0f);

	//  SPH force, F
	const int s = 1;
	for(int z=-s; z<=s; z++)
	for(int y=-s; y<=s; y++)
	for(int x=-s; x<=s; x++)
		addVel += compForceCell(gridPos + make_int3(x, y, z), index, pos, vel, oldPos, oldVel,
			pres, dens, pressure, density,	particleHash, cellStart);

	volatile uint si = particleHash[index].y;
	
	//  v = F*m*dt    a = F*m   v = a*dt
	addVel *= par.particleMass * par.timeStep;

	
	///  Height map  * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
	if (par.iHmap > 0)
	{	float3 vel3 = make_float3(vel) * -1;
		float rr = par.particleR + par.rotR;
		const int hz = 2,hy = 1;

		if (par.iHmap == 1)  // on XZ
		{		
			int ix = (pos.x -par.worldMin.x)/par.rotSpc;	float xf = ix*par.rotSpc +par.worldMin.x;
			int iz = (pos.z -par.worldMin.z)/par.rotSpc;	float zf = iz*par.rotSpc +par.worldMin.z;
		
			for(int j = -hz; j <= hz; j++)
			for(int i = -hz; i <= hz; i++)
			{
				float xh = xf + i*par.rotSpc;
				float zh = zf + j*par.rotSpc;
				for(int k = 0; k <= hy; k++)
				{
					float xn = xh / par.worldSizeD.x *PI2;
					float zn = zh / par.worldSizeD.z *PI2;
					
					float ss =	sinf(par.s1 * xn + par.s2*PI/180.f) *
								sinf(par.s3 * zn + par.s4*PI/180.f);
					if (par.s5 <= -1.f || ss > par.s5)
					{
						float yf = ss * par.hClose + par.r2Angle + par.worldMin.y;
						float yh = yf - k*par.rotSpc;
						addVel += collideSpheresR( make_float3(xh-pos.x, yh-pos.y, zh-pos.z), vel3, rr);
					}
				}
			}
		} else {	// on YZ  exactly as above, x <-> y
			int iy = (pos.y -par.worldMin.y)/par.rotSpc;	float yf = iy*par.rotSpc +par.worldMin.y;
			int iz = (pos.z -par.worldMin.z)/par.rotSpc;	float zf = iz*par.rotSpc +par.worldMin.z;
		
			for(int j = -hz; j <= hz; j++)
			for(int i = -hz; i <= hz; i++)
			{
				float yh = yf + i*par.rotSpc;
				float zh = zf + j*par.rotSpc;
				for(int k = 0; k <= hy; k++)
				{
					float yn = yh / par.worldSizeD.y *PI2;
					float zn = zh / par.worldSizeD.z *PI2;
					
					float ss =	sinf(par.s1 * yn + par.s2*PI/180.f) *
								sinf(par.s3 * zn + par.s4*PI/180.f);
					if (par.s5 <= -1.f || ss > par.s5)
					{
						float xf = ss * par.hClose + par.r2Angle + par.worldMin.x;
						float xh = xf - k*par.rotSpc;
						addVel += collideSpheresR( make_float3(xh-pos.x, yh-pos.y, zh-pos.z), vel3, rr);
					}
				}
			}
		}
	}

	///  Rotor  ///////////////////////////////////////////////////////////////
	if (par.rotType > 0)
	{	int sx = par.rotSize.x, sy = par.rotSize.y, sz = par.rotSize.z, cb = par.rotBlades;
		float r = par.rotR, sp = par.rotSpc, ca = PI2/cb, x2 = sx*0.5f, y2 = sy*0.5f;
		float rr = par.particleR + r;	float3 vel3 = make_float3(vel) * -1;
		
		switch (par.rotType)
		{
		case 1:		// pump rotor Z -
			if (pos.z > par.collPos.z -rr &&
				pos.z < par.collPos.z +rr + sp*sy)
			for(int z= 1; z<=sz; z++)
			for(int x= 0; x<=sx; x++)	for(int c= 0; c<cb; c++)
			{
				float a = -par.rAngle + c*ca+ z*par.rTwist, cs = cosf(a)*sp, sn = -sinf(a)*sp;
				for(int y= 0; y<=sy; y++)	//..if bnd xy
				{
					float4 pc = make_float4((x-x2)*cs - z*sn, (x-x2)*sn + z*cs, sp*y, 0);
						addVel += collideSpheresR( make_float3(par.collPos + pc - pos), vel3, rr);
				}
			}	break;

		case 2:		// pump rotor Y |
			if (pos.y > par.collPos.y -rr &&
				pos.y < par.collPos.y +rr + sp*sy)
			for(int z= 1; z<=sz; z++)
			for(int x= 0; x<=sx; x++)	for(int c= 0; c<cb; c++)
			{
				float a = -par.rAngle + c*ca+ z*par.rTwist, cs = cosf(a)*sp, sn = -sinf(a)*sp;
				for(int y= 0; y<=sy; y++)
				{
					float4 pc = make_float4((x-x2)*cs - z*sn, sp*y, (x-x2)*sn + z*cs, 0);
						addVel += collideSpheresR( make_float3(par.collPos + pc - pos), vel3, rr);
				}
			}	break;

		case 3:		// propeller Z -
			if (pos.z > par.collPos.z -rr - sp*sz/2.f &&
				pos.z < par.collPos.z +rr + sp*sz/2.f)
			{
			float4 rotPos = par.collPos;
			float aa = par.rAngle, tw = par.rTwist;
			if (par.r2Dist > 0.f)
				if (pos.x > 0.f) {  rotPos.x += par.r2Dist*0.5f;  }
				else			 {  rotPos.x -= par.r2Dist*0.5f;  aa = par.r2Angle;  tw *= par.r2twist;  }
				
			for(int c= 0; c<cb; c++)	{	float a = aa + c*ca;	// h = d-  x = r|
			for(int h= 0; h<=sy; h++)
			{	float dh = 0;  if (h==sy-1) dh=0.5f; else if (h==sy)  dh=1.2f;
				for(int x= 0; x<=sz; x++)
				{		float d = dh;  if (x==sz && d==0) d=0.4f;
						float k = cosf( (x-x2)*0.2f + PI*0.6f);  //d modif -^.
						float at = tw*(d*-0.05f -(h-y2)*k);	
					float ac = a + at, cs = cosf(ac)*sp, sn = -sinf(ac)*sp;
					float4 pc = make_float4(x*cs, x*sn, sp*(-d+h-y2) *k, 0);
					r = par.rotR*abs(1-d);
					addVel += collideSpheresR( make_float3(rotPos + pc - pos), vel3, par.particleR + r);
				}
			}	}
			}	break;
		}
	}
	///  sphere collider
	else
		addVel += collideSpheres(par.collPos - pos, -vel, par.particleR + par.collR);


	///  Accelerators  *************************************
	for (int i=0; i < NumAcc; i++)
	{	Accel& ac = par.acc[i];
		if (ac.type != ACC_Off)
		{
			float3 rel = make_float3(pos) - ac.pos;
			switch (ac.type)
			{
				case ACC_Box:	if (abs(rel.x) < ac.size.x && 
						abs(rel.y) < ac.size.y && abs(rel.z) < ac.size.z)
						addVel += ac.acc * par.timeStep;	break;
			
				case ACC_CylY:	{	float r = length(make_float2(rel.x/ac.size.x, rel.z/ac.size.z));
					if (abs(rel.y) < ac.size.y && r < 1.f)
						//rel.x*rel.x < ac.size.x*ac.size.x && rel.z*rel.z < ac.size.z*ac.size.z)
						addVel += ac.acc * par.timeStep;	}	break;
			
				case ACC_CylYsm: {	float r = length(make_float2(rel.x, rel.z));
					if (abs(rel.y) < ac.size.y && r < ac.size.x)
						addVel += ac.acc * (1.f - r/ac.size.z) * par.timeStep;  }  break;
		}	}
	}

	//  add new vel
	newVel[si] = vel + make_float4(addVel, 0.0f);


	///  coloring  . . . . . . . . . . . . . . . . . . . . . . . . . . . . 
	float3 color = make_float3(0.2, 0.5, 1);  float intens = 0.f;
	
	switch (par.clrType)
	{	case CLR_None:  return;
		
		case CLR_VelAcc:
		{	float v = 2.5f * length(make_float3(vel));
			float f = 0.02f * length(addVel) / par.timeStep;
				float clrV = par.brightness + par.contrast * v;
				float clrF = par.contrast * f;
			color = color * clrV + make_float3(0.7f, 0.35f, 0) * clrF;	}	break;

		case CLR_DensAcc:
		{	float d = 4.f* (dens - par.restDensity) / par.restDensity;
			float f = 0.02f * length(addVel) / par.timeStep;
				float clrD = par.brightness + par.contrast * d;
				float clrF = par.contrast * f;
			color = color * clrD + make_float3(0.7f, 0.7f, 0) * clrF;	}	break;
		
		case CLR_Vel:
		{	float v = 2.5f * length(make_float3(vel));
				float clrV = par.brightness + par.contrast * v;		intens = clrV;

	//* Foam implementation
    float4 NvelDif = make_float4(0.0f); 
	float4 NposDif = make_float4(0.0f); 
	float minvThreshold = 5.0f;
	float maxvThreshold = 20.0f;
	float vdiff = 0.0f; 
	float TrappedAirPot = 0.0f;
	uint gridHash = calcGridHash(gridPos);
	uint bucketStart = FETCH(cellStart, gridHash);
	float h = 3.0f;
    float3 Normal = make_float3(0);
	 
	 for(int x=-1;x<2;x++){
	 for(int y=-1;y<2;y++){
	 for(int z=-1;z<2;z++){
	//  iterate over particles in this cell
	for (uint i=0; i < par.maxParInCell; i++)
	{
	 	uint gridHash = calcGridHash(make_int3(gridPos.x+x,gridPos.y+y,gridPos.z+z));
    	uint bucketStart = FETCH(cellStart, gridHash);   
		uint index2 = bucketStart + i;
		uint2 cellData = FETCH(particleHash, index2);
		if (cellData.x != gridHash)  break;

		if (index2 != index)
		{
			float4 vel2 = FETCH(oldVel, index2); 
			float4 pos2 = FETCH(oldPos, index2);
			NvelDif = normalize(vel+vel2); //nvlog error when '-' ??
			NposDif = normalize(pos+pos2); //nvlog error when '-' ??
			vdiff += sqrt(dot(vel-vel2,vel-vel2)) * (1.0f-dot(NvelDif,NposDif)) * W(sqrt(dot(pos-pos2,pos-pos2)),h);

			/*
			Normal.x += (pos.y-pos2.y)*(pos.z-pos2.z);
			Normal.y += (pos.x-pos2.x)*(pos.z-pos2.z);
			Normal.z += (pos.y-pos2.y)*(pos.x-pos2.x);
			*/
		}
	}	
	}}}

	TrappedAirPot = phiClamp(vdiff,minvThreshold,maxvThreshold);

	/*
	float3 Normal;
	float4 pos2 = FETCH(oldPos, index+1); 
	float4 pos3 = FETCH(oldPos, index-1);
	float4 U = pos2-pos;
	float4 V = pos3-pos;

	Normal.x = U.y*V.z - U.z*V.y;
	Normal.y = U.z*V.x - U.x-V.z;
	Normal.z = U.x*V.y - U.y*V.x;
	*/
	float Iwc = 0;

	float minkThreshold = 0.1f;
	float maxkThreshold = 1.0f;
	float kEnergy = par.particleMass*1000*dot(vel,vel)*0.5;
	float kPot = phiClamp(kEnergy,minkThreshold,maxkThreshold);

	float Kta = 3.0f;
	float Kwc = 3.0f;
	float Nd = kPot*(Kta*TrappedAirPot+Kwc*Iwc);

	//color *= clrV;

    color = make_float3(Nd); 
	
	}	break;

//*End of modifications
		case CLR_VelRGB:
		{	color = make_float3(0.5f, 0.5f, 0.5f) + par.contrast * 2.5f * make_float3(vel);	}	break;

		case CLR_Accel:
		//{	float f = 0.000005f / par.particleMass * length(addVel) / par.timeStep;  // =force-  v = F*m*dt
		{	float f = 0.02f * length(addVel) / par.timeStep;	//  = accel   v = a*dt
				float clrF = par.brightness + par.contrast * f;		intens = clrF;
			color *= clrF;  }	break;

		case CLR_Dens:
		{	float d = 4.f* (dens - par.restDensity) / par.restDensity;
				float clrD = par.brightness + par.contrast * d;		intens = clrD;
			color *= clrD;	}	break;
	}
	if (par.iHue == 1)	{  color.x = intens;/*Hue*/  color.y = 0.f;  }

	
	///  dye  ` ` ` ` ` `
	if (par.dyeClear > 0)  dyeColor[si] = 0.f;
	else if (par.dyeType > 0)
	{
		float dyeCl = FETCH(dyeColor, si);  // intensity

		//  set clr
		float3 rel = make_float3(pos) - par.dyePos;
		switch (par.dyeType)
		{
		case 1:  // box
		{	float3& dsize = par.dyeSize;
			if (abs(rel.x) < dsize.x && abs(rel.y) < dsize.y && abs(rel.z) < dsize.z)	dyeCl = 1.f;  }  break;
		case 2:  // sphere
			if (length(rel) < par.dyeSize.y)	dyeCl = 1.f;  break;
		}

		//  fade
		dyeCl -= par.timeStep * par.dyeFade;
		if (dyeCl < 0.f)  dyeCl = 0.f;//
		if (dyeCl >= 0.f)
			dyeColor[si] = dyeCl;

		if (par.iHue == 0)
			color += make_float3(0.9f,0.9f,1)* dyeCl;
		else
			color.y = dyeCl;
	}
	clr[si] = make_float4(color, 1.f);
}
// END OF KERNEL.CU!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

extern "C"
{

//  Utility, data
#define CU  CUDA_SAFE_CALL

#define cuMapVbo(pv,vbo)		CU(cudaGLMapBufferObject((void**)&pv, vbo));
#define cuUnMapVbo(vbo)			CU(cudaGLUnmapBufferObject(vbo));

#define cuBindTex(tex,pv,size)	CU(hipBindTexture(0, tex, pv, size));
#define cuUnbindTex(tex)		CU(hipUnbindTexture(tex));


//void cudaInit(int argc, char **argv)	{   CUT_DEVICE_INIT(argc, argv);	}
/*bool cudaInitB(int argc, char **argv, bool showInfo)	//  no exit + info
{
	int deviceCount;
	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0)  {
		fprintf(stderr, "Error: no devices supporting CUDA.\n");
		return false;  }  //exit(EXIT_FAILURE);
	int dev = 0;
	cutGetCmdLineArgumenti(argc, (const char**)argv, "device", &dev);
	if (dev > deviceCount-1) dev = deviceCount - 1;
	hipDeviceProp_t p;
	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&p, dev));
	if (p.major < 1)  {
		fprintf(stderr, "Error: device does not support CUDA.\n");
		return false;	}
    if (dev == 0)  {
        if (p.major == 9999 && p.minor == 9999)
            printf("There is no device supporting CUDA.\n");
        else  if (deviceCount != 1)
            printf("There are %d devices supporting CUDA\n", deviceCount);  }
	
	if (cutCheckCmdLineFlag(argc, (const char**)argv, "quiet") == CUTFalse)
		fprintf(stderr, "Using device %d: %s\n\n", dev, p.name);
		
	if (showInfo)
	{
        printf("CUDA version:  %d.%d\n", p.major, p.minor);
        printf("Total memory:  %u bytes  (%u MB)\n", p.totalGlobalMem, p.totalGlobalMem/1024/1024);
		#if CUDART_VERSION >= 2000
        printf("Multiprocessors:  %d  Cores:  %d\n", p.multiProcessorCount, 8*p.multiProcessorCount);
		#endif
		printf("Clock rate:  %.4f GHz\n", p.clockRate * 1e-6f);
		printf(" Warp size:  %d\n", p.warpSize);
		printf(" Constant memory:  %u bytes\n", p.totalConstMem); 
		printf("  Shared memory per block:  %u bytes\n", p.sharedMemPerBlock);
		printf("  Registers per block:   %d\n", p.regsPerBlock);
		printf("  Max threads per block: %d\n", p.maxThreadsPerBlock);
		printf("  Max sizes of a block:  %d x %d x %d\n", p.maxThreadsDim[0], p.maxThreadsDim[1], p.maxThreadsDim[2]);
		printf("  Max sizes of a grid:   %d x %d x %d\n", p.maxGridSize[0], p.maxGridSize[1], p.maxGridSize[2]);
		printf("  Max memory pitch:    %u bytes\n", p.memPitch);
		printf("  Texture alignment:   %u bytes\n", p.textureAlignment);
	    #if CUDART_VERSION >= 2000
        printf("  Concurrent copy and execution:  %s\n\n", p.deviceOverlap ? "Yes" : "No");
		#endif
	}
	
	CUDA_SAFE_CALL(hipSetDevice(dev));
	return true;
}*/
void threadSync()						{	CU(hipDeviceSynchronize());	}


void allocateArray(void **devPtr, size_t size)	{	CU(hipMalloc(devPtr, size));	}
void freeArray(void *devPtr)					{	CU(hipFree(devPtr));	}

void registerGLvbo(uint vbo)		{	CU(cudaGLRegisterBufferObject(vbo));	}
void unregGLvbo(uint vbo)			{	CU(cudaGLUnregisterBufferObject(vbo));	}


void copyFromDevice(void* host, const void* device, unsigned int vbo, int size)
{
	if (vbo)  cuMapVbo(device, vbo);
	CU(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
	if (vbo)  cuUnMapVbo(vbo);
}
void copyToDevice(void* device, const void* host, int offset, int size)
{
	CU(hipMemcpy((char*) device + offset, host, size, hipMemcpyHostToDevice));
}

void setParameters(SimParams *hostParams)
{	// copy parameters to constant memory
	CU(hipMemcpyToSymbol(HIP_SYMBOL(par), hostParams, sizeof(SimParams)));
}


//  Round a / b to nearest higher integer value
int iDivUp(int a, int b) {	return a%b != 0 ? a/b+1 : a/b;	}

//  compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
	numThreads = min(blockSize, n);  numBlocks = iDivUp(n, numThreads);
}
//----------------------------------------------------------------------------------



///  integrate
void integrate(uint vboOldPos, uint vboNewPos, float4* oldVel, float4* newVel, int numParticles)
{
	int numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);

	float4 *oldPos, *newPos;
	cuMapVbo(oldPos, vboOldPos);  cuMapVbo(newPos, vboNewPos);

	integrateD<<< numBlocks, numThreads >>>(newPos, newVel, oldPos, oldVel);

	CUT_CHECK_ERROR("Kernel execution failed: Integrate");
	cuUnMapVbo(vboOldPos);  cuUnMapVbo(vboNewPos);

	/***/threadSync();
}


///  calcHash
void calcHash(uint vboPos, uint2* particleHash, int numParticles)
{
	int numThreads, numBlocks;
	computeGridSize(numParticles, 2*256, numBlocks, numThreads);

	float4 *pos;
	cuMapVbo(pos, vboPos);

	calcHashD<<< numBlocks, numThreads >>>(pos, particleHash);

	CUT_CHECK_ERROR("Kernel execution failed: calcHash");
	cuUnMapVbo(vboPos);

	/***/threadSync();
}


///  reorder
void reorder(uint vboOldPos, float4* oldVel, float4* sortedPos, float4* sortedVel,
		uint2* particleHash, uint* cellStart,  uint numParticles, uint numCells)
{
	int numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);
	CU(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

	float4 *oldPos;
	cuMapVbo(oldPos, vboOldPos);
	#if USE_TEX
	uint spar4 = numParticles*sizeof(float4);
	cuBindTex(oldPosTex, oldPos, spar4);	cuBindTex(oldVelTex, oldVel, spar4);
	#endif

	reorderD<<< numBlocks, numThreads >>>(particleHash, cellStart,
		oldPos, oldVel, sortedPos, sortedVel);

	CUT_CHECK_ERROR("Kernel execution failed: reorder");
	#if USE_TEX
	cuUnbindTex(oldPosTex);  cuUnbindTex(oldVelTex);
	#endif
	cuUnMapVbo(vboOldPos);

	/***/threadSync();
}


///  collide
void collide(uint timer,  uint vboOldPos, uint vboNewPos, /**/uint vboCLR2,
		float4* sortedPos, float4* sortedVel, float4* oldVel, float4* newVel,
		float* pressure, float* density, float* dyeColor,
		uint2* particleHash, uint* cellStart, uint numParticles, uint numCells)
{
//	/**/CU(cutStartTimer(timer));
	float4 *oldPos, *newPos, *clr2;
	cuMapVbo(clr2, vboCLR2);
	cuMapVbo(oldPos, vboOldPos);  cuMapVbo(newPos, vboNewPos);  

	#if USE_TEX
	uint spar4 = numParticles*sizeof(float4), spar = numParticles*sizeof(float);
	cuBindTex(oldPosTex, sortedPos, spar4);  cuBindTex(pressureTex, pressure, spar);
	cuBindTex(oldVelTex, sortedVel, spar4);	 cuBindTex(densityTex, density, spar);  cuBindTex(dyeColorTex, dyeColor, spar);//
	cuBindTex(particleHashTex, particleHash, numParticles*sizeof(uint2));
	cuBindTex(cellStartTex, cellStart, numCells*sizeof(uint));
	#endif


	int numThreads, numBlocks;
	computeGridSize(numParticles, 64, numBlocks, numThreads);
	

	computeDensityD<<< numBlocks, numThreads >>>(clr2, sortedPos,  pressure, density,  particleHash, cellStart);
	/***/threadSync();
//	/**/CU(cutStopTimer(timer));

	  computeForceD<<< numBlocks, numThreads >>>(newPos, newVel, sortedPos, sortedVel,  clr2, pressure, density, dyeColor/**/, particleHash, cellStart);
		//collideD<<< numBlocks, numThreads >>>(newPos, newVel, sortedPos, sortedVel, /**/clr2,  particleHash, cellStart);

	CUT_CHECK_ERROR("Kernel execution failed: Collide");

	cuUnMapVbo(vboNewPos);  cuUnMapVbo(vboOldPos);  cuUnMapVbo(vboCLR2);
	#if USE_TEX
	cuUnbindTex(oldPosTex);		cuUnbindTex(oldVelTex);
	cuUnbindTex(pressureTex)	cuUnbindTex(densityTex);	cuUnbindTex(dyeColorTex);//
	cuUnbindTex(particleHashTex);	cuUnbindTex(cellStartTex);
	#endif
	/***/threadSync();
}


}  //extern "C"